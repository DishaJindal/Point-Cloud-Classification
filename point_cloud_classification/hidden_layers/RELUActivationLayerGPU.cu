#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "../utilities/kernels.h"
#include "../utilities/utils.h"
#include "layer.h"
#include "RELUActivationLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {
	class RELUActivationLayerGPU : public RELUActivationLayer {
		RELUActivationLayerGPU() {};

		RELUActivationLayerGPU(int inputDim, int outputDim, int batchDim, bool lastLayer) {
			RELUActivationLayer(inputDim, outputDim, batchDim, lastLayer);

		}

		void forward(float *inputArg, float *outputArg, bool test) {

		}

		void backward(float *incomingGradient, float *outgoingGradient, float learningRate) {

		}
	};
}
