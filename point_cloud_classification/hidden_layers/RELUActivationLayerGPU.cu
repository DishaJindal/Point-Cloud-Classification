#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../common.h"
#include "../utilities/kernels.h"
#include "../utilities/utils.h"
#include "../utilities/matrix.h"
#include "layer.h"
#include "RELUActivationLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

__global__ void reluBackward(float* input, float* Z, int n, float* output) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		output[index] = (Z[index] > 0) ? input[index] : 0;
	}
}

namespace PointCloudClassification {
	class RELUActivationLayerGPU : public RELUActivationLayer {
		public:
		RELUActivationLayerGPU() {};

		RELUActivationLayerGPU(int inputDim, int batchDim, bool lastLayer) : RELUActivationLayer(inputDim, inputDim, batchDim, lastLayer) {

		}
		std::vector<float*> RELUActivationLayer::forward(std::vector<float*> inputArg, bool test) {
			float* flattenedInput;
			hipMalloc((void**)&flattenedInput, batchDim * inputDim * sizeof(float));
			int i = 0;
			for (auto current : inputArg) {
				hipMemcpy(flattenedInput + (i * inputDim), current, inputDim * sizeof(float), hipMemcpyDeviceToDevice);
				i++;
			}
			float* flattenedOutput;
			hipMalloc((void**)&flattenedOutput, batchDim * outputDim * sizeof(float));

			MatrixGPU* m = new MatrixGPU();
			m->ReluForward(flattenedInput, batchDim, inputDim, flattenedOutput);

			std::vector<float*> outputArg;
			for (int i = 0; i < batchDim; i++) {
				outputArg.push_back(flattenedOutput + (i * outputDim));
				Z.push_back(flattenedInput + (i * inputDim));
			}
			//free(flattenedOutput);

			return outputArg;
		}

		std::vector<float*> backward(std::vector<float*> incomingGradient, float learningRate) {
			std::vector<float*> outgoingGradient;
			for (int i = 0; i < batchDim; i++) {
				float* temp;
				hipMalloc((void**)&temp, inputDim * sizeof(float));
				dim3 fullBlocksPerGrid((inputDim + blockSize - 1) / blockSize);
				reluBackward << <fullBlocksPerGrid, blockSize >> > (incomingGradient[i], Z[i], inputDim, temp);
				outgoingGradient.push_back(temp);
			}
		}
	};
}
