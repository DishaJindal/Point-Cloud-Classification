#pragma once

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../common.h"
#include "../utilities/kernels.h"
#include "../utilities/parameters.h"
#include "../utilities/utils.h"
#include "../utilities/matrix.h"
#include "layer.h"
#include "fullyConnectedLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {

	std::vector<float*> FullyConnectedLayerGPU::forward(std::vector<float*> inputArg, bool test) {
		int i = 0;
		for (auto current : inputArg) {
			hipMemcpy(flattenedInputForward + (i * inputDim), current, inputDim * sizeof(float), hipMemcpyDeviceToDevice);
			i++;
		}
		MatrixGPU* m = new MatrixGPU();
		m->multiply(flattenedInputForward, W, batchDim, inputDim, outputDim, flattenedOutputForward);
		//free(flattenedInput);

		// Store input and output of this layer
		hipMemcpy(A, flattenedInputForward, batchDim * inputDim * sizeof(float), hipMemcpyDeviceToDevice);

		std::vector<float*> outputArg;
		for (int i = 0; i < batchDim; i++) {
			//m->add((float *)(flattenedOutputForward + (i * outputDim)), B, 1, outputDim, (float *)(flattenedOutputForward + (i * outputDim)));
			outputArg.push_back(flattenedOutputForward + (i * outputDim));
		}
		//free(flattenedOutput);


		return outputArg;
	}

	std::vector<float*> FullyConnectedLayerGPU::backward(std::vector<float*> incomingGradient, float learningRate) {
			
			
		int i = 0;
		for (auto current : incomingGradient) {
			hipMemcpy(flattenedInputBackward + (i * outputDim), current, outputDim * sizeof(float), hipMemcpyDeviceToDevice);
			i++;
		}

		MatrixGPU* m = new MatrixGPU();

		// Compute gradient w.r.t weights
		float *ATranspose;
		hipMalloc((void**)&ATranspose, inputDim * batchDim * sizeof(float));
		m->transpose(A, batchDim, inputDim, ATranspose);
		m->multiply(ATranspose, flattenedInputBackward, inputDim, batchDim, outputDim, dW);
		hipFree(ATranspose);

		// Compute outgoingGradient (w.r.t. input)
		m->multiplyTranspose(flattenedInputBackward, W, batchDim, outputDim, inputDim, flattenedOutputBackward);

		//Update weight matrix
		//m->subtractWithFactor(W, dW, learningRate, inputDim, outputDim, W);
		m->linearCombination(W, dW, (1.0f - Parameters::lamba_reg),-1.0f*learningRate, inputDim, outputDim, W);

		//hipMalloc((void**)&ATranspose, outputDim * batchDim * sizeof(float));
		//m->sumAcrossDim1(flattenedInputBackward, batchDim, outputDim, ATranspose);
		//m->linearCombination(B, ATranspose, 1.0f, -1.0f*learningRate, 1, outputDim, B);
		//hipFree(ATranspose);


		std::vector<float*> outgoingGradient;
		for (int i = 0; i < batchDim; i++) {
			outgoingGradient.push_back(flattenedOutputBackward + (i * inputDim));
		}
		//free(flattenedOutput);

		return outgoingGradient;

	}
};

