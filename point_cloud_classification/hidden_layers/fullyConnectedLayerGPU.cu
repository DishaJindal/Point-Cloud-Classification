#pragma once

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../common.h"
#include "../utilities/kernels.h"
#include "../utilities/parameters.h"
#include "../utilities/utils.h"
#include "../utilities/matrix.h"
#include "layer.h"
#include "fullyConnectedLayer.h"
#include <fstream>
#include <string>
#include ""
#include <iostream>
#include <fstream>
#include <iomanip>    // Needed for stream modifiers fixed and setprecision
using namespace std;


#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {

	void FullyConnectedLayerGPU::saveModel(std::string file_name) {
		// Save Weights
		float* W_host = new float[inputDim * outputDim];
		hipMemcpy(W_host, W, inputDim * outputDim * sizeof(float), hipMemcpyDeviceToHost);
		ofstream out(file_name + "_W.txt");
		for (int i = 0; i < inputDim * outputDim; ++i) {
			out << W_host[i] << "\n";
		}
		out.close();

		// Save Bias
		float* B_host = new float[outputDim];
		hipMemcpy(B_host, B, outputDim * sizeof(float), hipMemcpyDeviceToHost);
		ofstream out2(file_name + "_B.txt");
		for (int i = 0; i < outputDim; ++i) {
			out2 << B_host[i] << "\n";
		}
		out2.close();
		//std::cout << "\n\nSAVING FC\n\n";
		//Utilities::printArrayGPU(W, 10);
		//Utilities::printArrayGPU(B, 10);
		//std::cout << "\n\nSAVING FC\n\n";
	}

	std::vector<float*> FullyConnectedLayerGPU::forward(std::vector<float*> inputArg, bool test) {
		int i = 0;
		for (auto current : inputArg) {
			hipMemcpy(flattenedInputForward + (i * inputDim), current, inputDim * sizeof(float), hipMemcpyDeviceToDevice);
			i++;
		}
		MatrixGPU* m = new MatrixGPU();
		m->multiply(flattenedInputForward, W, batchDim, inputDim, outputDim, flattenedOutputForward);
		//free(flattenedInput);

		// Store input and output of this layer
		hipMemcpy(A, flattenedInputForward, batchDim * inputDim * sizeof(float), hipMemcpyDeviceToDevice);

		std::vector<float*> outputArg;
		for (int i = 0; i < batchDim; i++) {
			//m->add((float *)(flattenedOutputForward + (i * outputDim)), B, 1, outputDim, (float *)(flattenedOutputForward + (i * outputDim)));
			outputArg.push_back(flattenedOutputForward + (i * outputDim));
		}
		//free(flattenedOutput);


		return outputArg;
	}

	std::vector<float*> FullyConnectedLayerGPU::backward(std::vector<float*> incomingGradient, float learningRate) {
			
			
		int i = 0;
		for (auto current : incomingGradient) {
			hipMemcpy(flattenedInputBackward + (i * outputDim), current, outputDim * sizeof(float), hipMemcpyDeviceToDevice);
			i++;
		}

		MatrixGPU* m = new MatrixGPU();

		// Compute gradient w.r.t weights
		float *ATranspose;
		hipMalloc((void**)&ATranspose, inputDim * batchDim * sizeof(float));
		m->transpose(A, batchDim, inputDim, ATranspose);
		m->multiply(ATranspose, flattenedInputBackward, inputDim, batchDim, outputDim, dW);
		hipFree(ATranspose);

		// Compute outgoingGradient (w.r.t. input)
		m->multiplyTranspose(flattenedInputBackward, W, batchDim, outputDim, inputDim, flattenedOutputBackward);

		//Update weight matrix
		//m->subtractWithFactor(W, dW, learningRate, inputDim, outputDim, W);
		m->linearCombination(W, dW, (1.0f - Parameters::lamba_reg),-1.0f*learningRate, inputDim, outputDim, W);

		//hipMalloc((void**)&ATranspose, outputDim * batchDim * sizeof(float));
		//m->sumAcrossDim1(flattenedInputBackward, batchDim, outputDim, ATranspose);
		//m->linearCombination(B, ATranspose, 1.0f, -1.0f*learningRate, 1, outputDim, B);
		//hipFree(ATranspose);


		std::vector<float*> outgoingGradient;
		for (int i = 0; i < batchDim; i++) {
			outgoingGradient.push_back(flattenedOutputBackward + (i * inputDim));
		}
		//free(flattenedOutput);

		return outgoingGradient;

	}
};

