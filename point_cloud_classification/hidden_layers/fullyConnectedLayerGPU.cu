#pragma once

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../common.h"
#include "../utilities/kernels.h"
#include "../utilities/utils.h"
#include "../utilities/matrix.h"
#include "layer.h"
#include "fullyConnectedLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {

	class FullyConnectedLayerGPU : public FullyConnectedLayer {
		FullyConnectedLayerGPU() {};

		float* flattenedInputForward;
		float* flattenedOutputForward;

		float* flattenedInputBackward;
		float* flattenedOutputBackward;
		
	public:
		FullyConnectedLayerGPU(int inputDim, int outputDim, int batchDim, bool lastLayer) : FullyConnectedLayer(inputDim, outputDim, batchDim, lastLayer)  {
			hipMalloc((void **)&W, inputDim * outputDim * sizeof(float));
			float *weightRand = new float[inputDim * outputDim];
			Utilities::genArray(inputDim * outputDim, weightRand);
			hipMemcpy(W, weightRand, inputDim * outputDim * sizeof(float), hipMemcpyHostToDevice);
			hipMalloc((void**)&A, inputDim * batchDim * sizeof(float));
			hipMalloc((void**)&dW, inputDim * outputDim * sizeof(float));

			
			hipMalloc((void**)&flattenedInputForward, batchDim * inputDim * sizeof(float));
			hipMalloc((void**)&flattenedOutputForward, batchDim * outputDim * sizeof(float));
			hipMalloc((void**)&flattenedInputBackward, batchDim * outputDim * sizeof(float));
			hipMalloc((void**)&flattenedOutputBackward, batchDim * inputDim * sizeof(float));
		}


		std::vector<float*> FullyConnectedLayer::forward(std::vector<float*> inputArg, bool test) {
			int i = 0;
			for (auto current : inputArg) {
				hipMemcpy(flattenedInputForward + (i * inputDim), current, inputDim * sizeof(float), hipMemcpyDeviceToDevice);
				i++;
			}
			MatrixGPU* m = new MatrixGPU();
			m->multiply(flattenedInputForward, W, batchDim, inputDim, outputDim, flattenedOutputForward);
			//free(flattenedInput);

			// Store input and output of this layer
			hipMemcpy(A, flattenedInputForward, batchDim * inputDim * sizeof(float), hipMemcpyDeviceToDevice);

			std::vector<float*> outputArg;
			for (int i = 0; i < batchDim; i++) {
				outputArg.push_back(flattenedOutputForward + (i * outputDim));
			}
			//free(flattenedOutput);

			return outputArg;
		}

		std::vector<float*> backward(std::vector<float*> incomingGradient, float learningRate) {
			
			
			int i = 0;
			for (auto current : incomingGradient) {
				hipMemcpy(flattenedInputBackward + (i * outputDim), current, outputDim * sizeof(float), hipMemcpyDeviceToDevice);
				i++;
			}

			MatrixGPU* m = new MatrixGPU();

			// Compute gradient w.r.t weights
			float *ATranspose;
			hipMalloc((void**)&ATranspose, inputDim * batchDim * sizeof(float));
			m->transpose(A, batchDim, inputDim, ATranspose);
			m->multiply(ATranspose, flattenedInputBackward, inputDim, batchDim, outputDim, dW);
			hipFree(ATranspose);

			// Compute outgoingGradient (w.r.t. input)
			m->multiplyTranspose(flattenedInputBackward, W, batchDim, outputDim, inputDim, flattenedOutputBackward);

			//Update weight matrix
			m->subtractWithFactor(W, dW, learningRate, inputDim, outputDim, W);
			
			std::vector<float*> outgoingGradient;
			for (int i = 0; i < batchDim; i++) {
				outgoingGradient.push_back(flattenedOutputBackward + (i * inputDim));
			}
			//free(flattenedOutput);

			return outgoingGradient;

		}
	};
}
