#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "../utilities/kernels.h"
#include "../utilities/utils.h"
#include "layer.h"
#include "globalPoolingLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {
 
	class GlobalPoolingLayerGPU : public GlobalPoolingLayer {
		GlobalPoolingLayerGPU() {};
		
	public:
		GlobalPoolingLayerGPU(int inputDim, int outputDim, int batchDim, bool lastLayer) {
			GlobalPoolingLayer(inputDim, outputDim, batchDim, lastLayer);
		}


		void forward(float *inputArg, float *outputArg, bool test) {
			
		}

		void backward(float *incomingGradient, float *outgoingGradient, float learningRate) {

			
		}
	};
}
