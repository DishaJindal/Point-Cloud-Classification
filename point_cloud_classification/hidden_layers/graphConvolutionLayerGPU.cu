#pragma once

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../common.h"
#include "../utilities/kernels.h"
#include "../utilities/matrix.h"
#include "layer.h"
#include "graphConvolutionLayer.h"
#include <fstream>
#include <string>
#include ""
#include <chrono>
using namespace std::chrono;
#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128
#define MAX_STREAMS 16 

namespace PointCloudClassification {

	float* GraphConvolutionLayerGPU::get_chebeshev_polynomial(float* Tk1, float* Tk2, float* L, float mul) {
		MatrixGPU* m = new MatrixGPU();

		float* Tk;
		if (mul) {
			float* t;
			hipMalloc((void**)&t, numPoints * inputDim * sizeof(float));
			m->multiply(L, Tk1, numPoints, numPoints, inputDim, t);

			hipMalloc((void**)&Tk, numPoints * inputDim * sizeof(float));
			m->linearCombination(t, Tk2, 2, -1, numPoints, inputDim, Tk);
			hipFree(t);
		}
		else {
			float* t;
			hipMalloc((void**)&t, numPoints * numPoints * sizeof(float));
			m->multiply(L, Tk1, numPoints, numPoints, numPoints, t);

			hipMalloc((void**)&Tk, numPoints * numPoints * sizeof(float));
			m->linearCombination(t, Tk2, 2, -1, numPoints, numPoints, Tk);
			hipFree(t);
		}
		return Tk;
	}

	void GraphConvolutionLayerGPU::forwardUtil(float* current_input, float* current_L, float* outp, hipStream_t stream) {
		MatrixGPU* m = new MatrixGPU();
		float* Tk;

		// Tk_minus_1 and Tk_minus_2
		Tk_minus_2 = current_input;
		m->multiply(current_L, current_input, numPoints, numPoints, inputDim, Tk_minus_1, stream);

		for (int k = 0; k < numFilters; k++) {
			if (k == 0) {
				Tk = Tk_minus_2;
			}
			else if (k == 1) {
				Tk = Tk_minus_1;
			}
			else {
				Tk = get_chebeshev_polynomial(Tk_minus_1, Tk_minus_2, current_L, true);
				Tk_minus_2 = Tk_minus_1;
				Tk_minus_1 = Tk;
			}

			if (k == 0) {
				m->multiply(Tk, theta[k], numPoints, inputDim, outputDim, outp, stream);
			}
			else {
				float* temp_out;
				hipMalloc((void**)&temp_out, numPoints * outputDim * sizeof(float));
				m->multiply(Tk, theta[k], numPoints, inputDim, outputDim, temp_out, stream);
				m->add(outp, temp_out, numPoints, outputDim, outp);
				hipFree(temp_out);
			}
		}
		m->linearCombination(outp, outp, (1.0f / numFilters), 0, numPoints, outputDim, outp);
	}
	std::vector<float*> GraphConvolutionLayerGPU::forward(std::vector<float*> inputArg, bool test) {
		int num_streams = batchDim;
		if (batchDim > MAX_STREAMS)
			num_streams = MAX_STREAMS;

		hipStream_t streams[MAX_STREAMS];
		for (int i = 0; i < num_streams; ++i) { hipStreamCreate(&streams[i]); }

		this->X = std::vector < float* >(inputArg.begin(), inputArg.begin() + batchDim);
		this->L = std::vector < float* >(inputArg.begin() + batchDim, inputArg.end());
		for (int i = 0; i < batchDim; i++) {
			float* current_input = inputArg[i];
			float* current_L = inputArg[i + batchDim];
			float* outp = output[i];
			forwardUtil(current_input, current_L, outp, streams[i%MAX_STREAMS]);
		}

		for (int i = 0; i < num_streams; ++i)
		{
			hipStreamSynchronize(streams[i]);
			hipStreamDestroy(streams[i]);
		}
		return output;
	}

	std::vector<float*> GraphConvolutionLayerGPU::backward(std::vector<float*> incomingGradient, float learningRate) {
		std::vector<float*> outgoingGradient;
		MatrixGPU* m = new MatrixGPU();
		m->getIdentityMatrix(numPoints, Tk_minus_2_back);
		float* Tk;
		int number_of_samples = incomingGradient.size();
		for (int i = 0; i < number_of_samples; i++) {
			float* current_L = L[i];
			float* current_input = X[i];
			float* current_gradient = incomingGradient[i];

			Tk_minus_1_back = current_L;

			for (int k = 0; k < numFilters; k++) {
				//auto start = high_resolution_clock::now();
				if (k == 0) {
					Tk = Tk_minus_2_back;
				}
				else if (k == 1) {
					Tk = Tk_minus_1_back;
				}
				else {
					Tk = get_chebeshev_polynomial(Tk_minus_1_back, Tk_minus_2_back, current_L, false);
					Tk_minus_2_back = Tk_minus_1_back;
					Tk_minus_1_back = Tk;
				}

				// Update theta (weights) --> Local Gradient
				m->multiply(Tk, current_input, numPoints, numPoints, inputDim, TX);

				m->transpose(TX, numPoints, inputDim, TXT);

				m->multiply(TXT, current_gradient, inputDim, numPoints, outputDim, dtheta);

				m->linearCombination(theta[k], dtheta, 1, (-1.0f *learningRate) / numFilters, inputDim, outputDim, theta[k]);

				// Calculate outgoing gradient
				m->multiply(Tk, current_gradient, numPoints, numPoints, outputDim, TG);
				m->transpose(theta[k], inputDim, outputDim, thetaT);
				if (k == 0) {
					m->multiply(TG, thetaT, numPoints, outputDim, inputDim, outgoing_gradient[i]);
				}
				else {
					m->multiply(TG, thetaT, numPoints, outputDim, inputDim, temp);
					m->add(outgoing_gradient[i], temp, numPoints, inputDim, outgoing_gradient[i]);
				}
				//auto stop = high_resolution_clock::now();
				//auto duration = duration_cast<microseconds>(stop - start);
				//std::cout << "DEBUG Per Filter: " << duration.count() << " microseconds" << std::endl;
			}
			m->linearCombination(outgoing_gradient[i], outgoing_gradient[i], (1.0f / numFilters), 0, numPoints, inputDim, outgoing_gradient[i]);
		}
		return outgoing_gradient;
	}
};

