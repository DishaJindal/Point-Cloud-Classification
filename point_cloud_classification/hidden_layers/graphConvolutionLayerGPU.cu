#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "../utilities/kernels.h"
#include "../utilities/utils.h"
#include "layer.h"
#include "graphConvolutionLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {

	class GraphConvolutionLayerGPU : public GraphConvolutionLayer {
		GraphConvolutionLayerGPU() {};
		
	public:
		GraphConvolutionLayerGPU(int numPoints, int inputDim, int outputDim, int batchDim, int numFilters, bool lastLayer) {
			GraphConvolutionLayer(numPoints, inputDim, outputDim, batchDim, numFilters, lastLayer);
		}


		void forward(float *inputArg, float *outputArg, bool test) {
			
		}

		void backward(float *incomingGradient, float *outgoingGradient, float learningRate) {

			
		}
	};
}
