#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../common.h"
#include "../utilities/kernels.h"
#include "../utilities/utils.h"
#include "layer.h"
#include "sigmoidActivationLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {

	void sigmoidActivationLayerGPU::forward(float *inputArg, float *outputArg, bool test) {

	}

	void sigmoidActivationLayerGPU::backward(float *incomingGradient, float *outgoingGradient, float learningRate) {

	}
};

