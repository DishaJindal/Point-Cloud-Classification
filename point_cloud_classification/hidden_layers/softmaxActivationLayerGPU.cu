#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../common.h"
#include "../utilities/kernels.h"
#include "../utilities/utils.h"
#include "../utilities/matrix.h"
#include "layer.h"
#include "softmaxActivationLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

//__global__ void kernExp(float* input, int m, int n, float* output) {
//	int index = blockIdx.x * blockDim.x + threadIdx.x;
//	if (index < m * n) {
//		output[index] = exp(input[index]);
//	}
//}
//
//__global__ void kernDivideSum(float* input, float* sum, int m, int n, float* output) {
//	int index = blockIdx.x * blockDim.x + threadIdx.x;
//	int row = index / n;
//	if (index < m * n) {
//		output[index] = input[index] / sum[row];
//	}
//}

namespace PointCloudClassification {

	class softmaxActivationLayerGPU : public softmaxActivationLayer {
		softmaxActivationLayerGPU() {};
	public:
		softmaxActivationLayerGPU(int inputDim, int batchDim, bool lastLayer) : softmaxActivationLayer(inputDim, inputDim, batchDim, lastLayer) {
			
		}

		std::vector<float*> forward(std::vector<float*> inputArg, bool test) {
			float* flattenedInput;
			hipMalloc((void**)&flattenedInput, batchDim * inputDim * sizeof(float));
			int i = 0;
			for (auto current : inputArg) {
				hipMemcpy(flattenedInput + (i * inputDim), current, inputDim * sizeof(float), hipMemcpyDeviceToDevice);
				i++;
			}
			float* flattenedOutput;
			hipMalloc((void**)&flattenedOutput, batchDim * outputDim * sizeof(float));

			float* temp;
			hipMalloc((void**)&temp, batchDim * outputDim * sizeof(float));

			dim3 fullBlocksPerGrid((batchDim * inputDim + blockSize - 1) / blockSize);
			//kernExp <<<fullBlocksPerGrid, blockSize >>> (flattenedInput, batchDim, inputDim, temp);

			MatrixGPU* m = new MatrixGPU();
			float* tempT;
			hipMalloc((void**)&tempT, batchDim * outputDim * sizeof(float));
			m->transpose(temp, batchDim, outputDim, tempT);
			//hipFree(temp);

			float* sum;
			hipMalloc((void**)&sum, batchDim * outputDim * sizeof(float));
			m->meanAcrossDim1(tempT, outputDim, batchDim, sum); //CHANGE THIS TO SUM --> m->sumAcrossDim1(tempT, outputDim, batchDim, sum);

			//dim3 fullBlocksPerGrid((batchDim * inputDim + blockSize - 1) / blockSize);
			//kernDivideSum << <fullBlocksPerGrid, blockSize >> > (temp, sum, batchDim, inputDim, flattenedOutput);

			hipFree(temp);
			hipFree(tempT);
			hipFree(sum);

			std::vector<float*> outputArg;
			for (int i = 0; i < batchDim; i++) {
				outputArg.push_back(flattenedOutput + (i * outputDim));
			}

			return outputArg;
		}

		std::vector<float*> backward(std::vector<float*> incomingGradient, float learningRate) {
			std::vector<float*> sample;
			return sample;
		}
	};
}
