#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "../../utilities/kernels.h"
#include "../layer.h"
#include "../fullyConnectedLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	


	void genArray(int n, float *a) {
		srand(11);

		for (int i = 0; i < n; i++) {
			a[i] = ((2 *((rand() * 1.0 )/ RAND_MAX)) - 1) * 0.0002;
		}
	}

	class FullyConnectedLayerGPU : public FullyConnectedLayer {
		FullyConnectedLayerGPU() {};
		
	public:
		FullyConnectedLayerGPU(int inputDim, int outputDim, int batchDim, bool lastLayer) {
			FullyConnectedLayer(inputDim, outputDim, batchDim, lastLayer);
			hipMalloc((void **)&W, inputDim * outputDim * sizeof(float));
			float *weightRand = new float[inputDim * outputDim];
			genArray(inputDim * outputDim, weightRand);
			hipMemcpy(W, weightRand, inputDim * outputDim * sizeof(float), hipMemcpyHostToDevice);
			hipMalloc((void**)&A, inputDim * batchDim * sizeof(float));
		}


		void forward(float *inputArg, float *outputArg, bool test) {
			hipMemcpy(A, inputArg, batchDim * inputDim * sizeof(float), hipMemcpyDeviceToDevice);
			int gridRows = (batchDim*outputDim + blockSize - 1) / blockSize;

			Utilities::kernMultiplyMatrices<<<gridRows, blockSize >>>(inputArg, W, outputArg, batchDim, inputDim, outputDim);
			checkCUDAError("kernMultiplyMatrices");

			dim3 fullBlocksPerGrid((outputDim*batchDim + blockSize - 1) / blockSize);
			if (!lastLayer) {
				Utilities::kernActivateReLU << <fullBlocksPerGrid, blockSize >> > (outputArg, outputDim*batchDim);
			}
			else {
				float *output = new float[outputDim * batchDim];
				hipMemcpy(output, outputArg, batchDim * outputDim * sizeof(float), hipMemcpyDeviceToHost);
				float *softmaxDenominator = new float[batchDim];
				memset(softmaxDenominator, 0, batchDim * sizeof(float));
				for (int j = 0; j < batchDim; j++) {
					for (int i = 0; i < outputDim; i++) {
						softmaxDenominator[j] += exp(output[j * outputDim + i]);
					}
				}

				float *devSoftmaxDenominator;
				hipMalloc((void **)&devSoftmaxDenominator, batchDim * sizeof(float));
				hipMemcpy(devSoftmaxDenominator, softmaxDenominator, batchDim * sizeof(float), hipMemcpyHostToDevice);
				Utilities::kernActivateSoftmax << <fullBlocksPerGrid, blockSize >> > (outputArg, batchDim * outputDim, outputDim, devSoftmaxDenominator);
				checkCUDAError("kernActivateSoftmax");

				delete(output);

				hipFree(devSoftmaxDenominator);
				checkCUDAError("hipFree");
			}

			if (test) {
				printf("\n\n\tWeights : ");
				float *tempWeight = new float[inputDim * outputDim];
				hipMemcpy(tempWeight, W, inputDim * outputDim * sizeof(float), hipMemcpyDeviceToHost);
				for (int i = 0; i < inputDim * outputDim; i++) {
					if (i % outputDim == 0) {
						printf("\n\t\t");
					}
					printf("%f ", tempWeight[i]);
				}
				delete(tempWeight);
			}
		}

		void backward(float *incomingGradient, float *outgoingGradient, float learningRate) {

			float *weightTranspose;
			hipMalloc((void**)&weightTranspose, inputDim * outputDim * sizeof(float));
			checkCUDAError("hipMalloc");

			int gridRows = (inputDim*outputDim + blockSize - 1) / blockSize;
			Utilities::kernTransposeMatrices << <gridRows, blockSize >> > (W, weightTranspose, inputDim, outputDim);
			checkCUDAError("kernTransposeMatrices");

			float *outgoingGradientLocal;
			hipMalloc((void**)&outgoingGradientLocal, inputDim*batchDim * sizeof(float));
			checkCUDAError("hipMalloc");


			gridRows = (inputDim*batchDim + blockSize - 1) / blockSize;
			Utilities::kernMultiplyMatrices << <gridRows, blockSize >> > (incomingGradient, weightTranspose, outgoingGradientLocal, batchDim, outputDim, inputDim);
			checkCUDAError("kernMultiplyMatrices");

			hipFree(weightTranspose);
			checkCUDAError("hipFree");

			float *inputDerivatived;
			hipMalloc((void**)&inputDerivatived, batchDim * inputDim * sizeof(float));
			dim3 fullBlocksPerGrid((inputDim * batchDim + blockSize - 1) / blockSize);
			Utilities::kernActivateReLUDerivative << <fullBlocksPerGrid, blockSize >> > (A, inputDerivatived, inputDim * batchDim);
			checkCUDAError("kernActivateReLUDerivative");


			gridRows = (inputDim*batchDim + blockSize - 1) / blockSize;
			Utilities::kernMultMatricesHammard << <gridRows, blockSize >> > (outgoingGradientLocal, inputDerivatived, outgoingGradient, batchDim, inputDim);
			checkCUDAError("kernMultMatricesHammard");

			hipFree(inputDerivatived);
			checkCUDAError("hipFree");


			float *inputTranspose;
			hipMalloc((void**)&inputTranspose, inputDim * batchDim * sizeof(float));

			gridRows = (inputDim*batchDim + blockSize - 1) / blockSize;
			Utilities::kernTransposeMatrices << <gridRows, blockSize >> > (A, inputTranspose, batchDim, inputDim);
			checkCUDAError("kernTransposeMatrices");

			float *gradient;
			hipMalloc((void**)&gradient, inputDim * outputDim * sizeof(float));
			gridRows = (inputDim*outputDim + blockSize - 1) / blockSize;
			Utilities::kernMultiplyMatrices << <gridRows, blockSize >> > (inputTranspose, incomingGradient, gradient, inputDim, batchDim, outputDim);
			checkCUDAError("kernMultiplyMatrices");

			hipFree(inputTranspose);
			checkCUDAError("hipFree");


			Utilities::kernMultMatricesWithScalar << <gridRows, blockSize >> > (gradient, gradient, inputDim, outputDim, learningRate);
			checkCUDAError("kernMultMatricesWithScalar");


			Utilities::kernSubtractMatrices << <gridRows, blockSize >> > (W, gradient, W, inputDim, outputDim);
			checkCUDAError("kernSubtractMatrices");

			hipFree(gradient);
			checkCUDAError("hipFree");
		}
	};
}
