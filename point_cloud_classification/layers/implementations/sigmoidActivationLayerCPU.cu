#include "hip/hip_runtime.h"
#include "common.h"
#include "../../utilities/kernels.h"
#include "../layer.h"
#include "../sigmoidActivationLayer.h"
#include <fstream>
#include <string>

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	void genArray(int n, float *a) {
		srand(11);

		for (int i = 0; i < n; i++) {
			a[i] = ((2 *((rand() * 1.0 )/ RAND_MAX)) - 1) * 0.0002;
		}
	}

	class sigmoidActivationLayerCPU : public sigmoidActivationLayer {
		sigmoidActivationLayerCPU() {};

		sigmoidActivationLayerCPU(int inputDim, int outputDim, int batchDim, bool lastLayer) {
			sigmoidActivationLayer(inputDim, outputDim, batchDim, lastLayer);
		}

		private float sigmoid(float z){
			float sig = (1.0f / (1.0f + exp(-z)));
			return sig;
		}

		/*
			inputArg -> batchDim x inputDim
			outputArg -> batchDim x inputDim
		*/
		void forward(float *inputArg, float *outputArg, bool test) {
			for(int i = 0; i < batchDim; i++){
				for(int j = 0; j < inputDim; j++){
					outputArg[i * inputDim + j] = sigmoid(inputArg[i * inputDim + j]);
					A[i * inputDim + j] = outputArg[i * inputDim + j];
					Z[i * inputDim + j] = inputArg[i * inputDim + j];
				}
			}
		}

		void backward(float *incomingGradient, float *outgoingGradient, float learningRate) {
			for(int i = 0; i < batchDim; i++){
				for(int j = 0; j < inputDim; j++){
					outgoingGradient[i * inputDim + j] = incomingGradient[i * inputDim + j] * A[i * inputDim + j] * (1 - A[i * inputDim + j]);
				}
			}
		}
	};
}
