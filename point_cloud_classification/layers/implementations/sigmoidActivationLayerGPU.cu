#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "../../utilities/kernels.h"
#include "../layer.h"
#include "../sigmoidActivationLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

namespace PointCloudClassification {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	void genArray(int n, float *a) {
		srand(11);

		for (int i = 0; i < n; i++) {
			a[i] = ((2 *((rand() * 1.0 )/ RAND_MAX)) - 1) * 0.0002;
		}
	}

	class sigmoidActivationLayerGPU : public sigmoidActivationLayer {
		sigmoidActivationLayerGPU() {};

		sigmoidActivationLayerGPU(int inputDim, int outputDim, int batchDim, bool lastLayer) {
			sigmoidActivationLayer(inputDim, outputDim, batchDim, lastLayer);
			hipMalloc((void **)&weight, inputDim * outputDim * sizeof(float));
			float *weightRand = new float[inputDim * outputDim];
			genArray(inputDim * outputDim, weightRand);
			hipMemcpy(weight, weightRand, inputDim * outputDim * sizeof(float), hipMemcpyHostToDevice);
			hipMalloc((void**)&inputs, inputDim * batchDim * sizeof(float));
		}
	};
}
