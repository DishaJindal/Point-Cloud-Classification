#include "hip/hip_runtime.h"
#include "common.h"
#include "network.h"
#include "hidden_layers/layer.h"
#include "hidden_layers/loss.h"
#include "graph/graph.h"
#include "utilities/matrix.h"
#include "utilities/parameters.h"
#include "utilities/utils.h"
#include "../src/testing_helpers.hpp"
#include <fstream>
#include <string>
#include <chrono>

using namespace std::chrono;

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128
#define debug false
#define time false
#define graph_time true

namespace PointCloudClassification {

	NetworkCPU::NetworkCPU(int numClasses, int batchSize) {
		this->numClasses = numClasses;
		this->batchSize = batchSize;
	}

	void NetworkCPU::addLayer(Layer* layer){
		this->layers.push_back(layer);
	}

	void NetworkCPU::setLoss(Loss* loss) {
		this->loss = loss;
	}

	void NetworkCPU::buildArchitecture()
	{
		// GCN Layer 1
		PointCloudClassification::GraphConvolutionLayerCPU gcn_layer1(Parameters::num_points, Parameters::input_features, Parameters::gcn1_out_features, Parameters::batch_size, Parameters::chebyshev1_order, false);
		this->gcn_layer1 = gcn_layer1;
		
		// Dropout 1
		PointCloudClassification::DropoutLayerCPU dropout_layer1(Parameters::num_points, Parameters::gcn1_out_features, Parameters::batch_size, false, Parameters::keep_drop_prob1);
		this->dropout_layer1 = dropout_layer1;

		// Global Pooling 1
		PointCloudClassification::GlobalPoolingLayerCPU gp_layer1(Parameters::num_points, Parameters::gcn1_out_features, Parameters::batch_size, false);
		this->gp_layer1 = gp_layer1;

		// GCN Layer 2
		PointCloudClassification::GraphConvolutionLayerCPU gcn_layer2(Parameters::num_points, Parameters::gcn1_out_features, Parameters::gcn2_out_features, Parameters::batch_size, Parameters::chebyshev2_order, false);
		this->gcn_layer2 = gcn_layer2;

		// Dropout 2
		PointCloudClassification::DropoutLayerCPU dropout_layer2(Parameters::num_points, Parameters::gcn2_out_features, Parameters::batch_size, false, Parameters::keep_drop_prob2);
		this->dropout_layer2 = dropout_layer2;

		// Global Pooling 2
		PointCloudClassification::GlobalPoolingLayerCPU gp_layer2(Parameters::num_points, Parameters::gcn2_out_features, Parameters::batch_size, false);
		this->gp_layer2 = gp_layer2;

		// Concatenate GCN Layer 1 and GCN Layer 2


		// Dropout 3
		int cat_features = (Parameters::gcn1_out_features + Parameters::gcn2_out_features);
		PointCloudClassification::DropoutLayerCPU dropout_layer3(cat_features, 2, Parameters::batch_size, false, Parameters::keep_drop_prob3);
		this->dropout_layer3 = dropout_layer3;

		// Fully Connected Layer 1
		PointCloudClassification::FullyConnectedLayerCPU fc_layer1(cat_features * 2, Parameters::fc1_out_features, Parameters::batch_size, false);
		this->fc_layer1 = fc_layer1;

		// ReLU 1
		PointCloudClassification::RELUActivationLayerCPU relu1(Parameters::fc1_out_features, Parameters::batch_size, false);
		this->relu1 = relu1;

		// Dropout 4
		PointCloudClassification::DropoutLayerCPU dropout_layer4(Parameters::fc1_out_features, 1, Parameters::batch_size, false, Parameters::keep_drop_prob4);
		this->dropout_layer4 = dropout_layer4;

		// Fully Connected Layer 2
		PointCloudClassification::FullyConnectedLayerCPU fc_layer2(Parameters::fc1_out_features, Parameters::num_classes, Parameters::batch_size, false);
		this->fc_layer2 = fc_layer2;
	}

	std::vector<float*> NetworkCPU::forward(std::vector<float*> input, bool test) {
		std::vector<float*> output, temp1, temp2;
		auto start = high_resolution_clock::now();
		output = gcn_layer1.forward(input, false);
		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Graph Convolution Layer 1 Forward / batch) ==> " << duration.count()<< " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}
		if (debug) {
			std::cout << "############################# FORWARD #################################### \n";
			std::cout << "gcn1  " << std::endl;
			Utilities::printVectorOfFloats(output, 5);
		}
		start = high_resolution_clock::now();
		output = dropout_layer1.forward(output, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Dropout 1 Forward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}
		if (debug) {
			std::cout << "D 1 " << std::endl;
			Utilities::printVectorOfFloats(output, 5);
		}
		start = high_resolution_clock::now();
		temp1 = gp_layer1.forward(output, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Global Pooling 1 Forward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}
		if (debug) {
			std::cout << "GP 1 " << std::endl;
			Utilities::printVectorOfFloats(temp1, 5);
		}
		std::vector<float*> batch_L = std::vector<float*>(input.begin() + Parameters::batch_size, input.end());
		std::vector<float*> output_with_L;
		output_with_L.reserve(output.size() + batch_L.size()); // preallocate memory
		output_with_L.insert(output_with_L.end(), output.begin(), output.end());
		output_with_L.insert(output_with_L.end(), batch_L.begin(), batch_L.end());

		start = high_resolution_clock::now();
		temp2 = gcn_layer2.forward(output_with_L, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Graph Convolution Layer 2 Forward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "GCN2 " << std::endl;
			Utilities::printVectorOfFloats(temp2, 5);
		}
		start = high_resolution_clock::now();
		temp2 = dropout_layer2.forward(temp2, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Dropout 2 Forward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "D2 " << std::endl;
			Utilities::printVectorOfFloats(temp2, 5);
		}
		start = high_resolution_clock::now();
		temp2 = gp_layer2.forward(temp2, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Global Pooling 2 Forward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "GP 2 " << std::endl;
			Utilities::printVectorOfFloats(temp2, 5);
		}
		// Concatenate
		std::vector<float*> cat_vec;
		for (int i = 0; i < Parameters::batch_size; i++) {
			float* cat = (float*)malloc((Parameters::gcn1_out_features + Parameters::gcn2_out_features) * 2 *sizeof(float));
			std::copy(temp1[i], temp1[i] + Parameters::gcn1_out_features * 2, cat);
			std::copy(temp2[i], temp2[i] + Parameters::gcn2_out_features * 2, cat + Parameters::gcn1_out_features * 2);
			cat_vec.push_back(cat);
		}
		
		if (debug) {
			std::cout << "Cat " << std::endl;
			Utilities::printVectorOfFloats(cat_vec, 5);
		}
		start = high_resolution_clock::now();
		output = dropout_layer3.forward(cat_vec, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Dropout 3 Forward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "D 3 " << std::endl;
			Utilities::printVectorOfFloats(output, 50);
		}
		start = high_resolution_clock::now();
		output = fc_layer1.forward(output, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Fully Connected Layer 1 Forward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "FC 1 " << std::endl;
			Utilities::printVectorOfFloats(output, 5);
		}
		start = high_resolution_clock::now();
		output = relu1.forward(output, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (RELU 1 Forward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "R 1 " << std::endl;
			Utilities::printVectorOfFloats(output, 5);
		}
		start = high_resolution_clock::now();
		output = dropout_layer4.forward(output, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Dropout 4 Forward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "D 4 " << std::endl;
			Utilities::printVectorOfFloats(output, 5);
		}
		start = high_resolution_clock::now();
		output = fc_layer2.forward(output, false);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Fully Connected Layer 2 Forward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "FC 2 " << std::endl;
			Utilities::printVectorOfFloats(output, 5);
		}
		return output;
	}

	float NetworkCPU::calculateLoss(std::vector<float*> prediction, std::vector<float*> trueLabel) {
		return this->loss->cost(prediction, trueLabel);
	}

	void NetworkCPU::backward(std::vector<float*> prediction, std::vector<float*> trueLabel, float learningRate) {
		// Get the gradient of the loss
		auto start = high_resolution_clock::now();
		std::vector<float*> dloss = this->loss->dcost(prediction, trueLabel);
		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Loss gradient / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		std::vector<float*> incomingGradient(dloss);

		if (debug) {
			std::cout << "############################# BACKWARD #################################### \n";
			std::cout << "Loss " << std::endl;
			Utilities::printVectorOfFloats(incomingGradient, 5);
		}
		start = high_resolution_clock::now();
		incomingGradient = fc_layer2.backward(incomingGradient, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Fully Connected Layer 2 Backward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "FC2 " << std::endl;
			Utilities::printVectorOfFloats(incomingGradient, 5);
		}
		start = high_resolution_clock::now();
		incomingGradient = dropout_layer4.backward(incomingGradient, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Dropout 4 Backward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "Dropout 4 " << std::endl;
			Utilities::printVectorOfFloats(incomingGradient, 5);
		}
		start = high_resolution_clock::now();
		incomingGradient = relu1.backward(incomingGradient, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (RELU 1 Backward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "RELU 1 " << std::endl;
			Utilities::printVectorOfFloats(incomingGradient, 5);
		}
		start = high_resolution_clock::now();
		incomingGradient = fc_layer1.backward(incomingGradient, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Fully Connected Layer 1 Backward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "FC1 " << std::endl;
			Utilities::printVectorOfFloats(incomingGradient, 5);
		}
		start = high_resolution_clock::now();
		incomingGradient = dropout_layer3.backward(incomingGradient, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Dropout Layer 3 Backward / batch) ==> " << duration.count()  << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)")
		}
		if (debug) {
			std::cout << "Dropout 3 " << std::endl;
			Utilities::printVectorOfFloats(incomingGradient, 5);
		}
		// Split
		std::vector<float*> gp1, gp2;
		for (int i = 0; i < Parameters::batch_size; i++) {
			gp1.push_back(incomingGradient[i]);
			gp2.push_back(incomingGradient[i] + Parameters::gcn1_out_features * 2);
		}
		//std::cout << "Split " << std::endl;
		//Utilities::printVectorOfFloats(incomingGradient, 5);

		start = high_resolution_clock::now();
		gp1 = gp_layer2.backward(gp1, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Global Pooling Layer 2 Backward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "GP 2 " << std::endl;
			Utilities::printVectorOfFloats(gp1, 5);
		}
		start = high_resolution_clock::now();
		gp1 = dropout_layer2.backward(gp1, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Dropout Layer 2 Backward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "Dropout 2 " << std::endl;
			Utilities::printVectorOfFloats(gp1, 5);
		}
		start = high_resolution_clock::now();
		gp1 = gcn_layer2.backward(gp1, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Graph Convolution Layer 2 Backward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}
		if (debug) {
			std::cout << "GCN 2 " << std::endl;
			Utilities::printVectorOfFloats(gp1, 5);
		}

		start = high_resolution_clock::now();
		gp2 = gp_layer1.backward(gp2, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Global Pooling Layer 1 Backward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}
		if (debug) {
			std::cout << "GP 1 " << std::endl;
			Utilities::printVectorOfFloats(gp2, 5);
		}
		start = high_resolution_clock::now();
		gp2 = dropout_layer1.backward(gp2, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Dropout 1 Backward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}
		if (debug) {
			std::cout << "Dropout 1 " << std::endl;
			Utilities::printVectorOfFloats(gp2, 5);
		}
		// Add
		for (int i = 0; i < Parameters::batch_size; i++) {
			for (int j = 0; j < Parameters::num_points * Parameters::gcn1_out_features; j++) {
				gp2[i][j] += gp1[i][j];
			}
		}
		if (debug) {
			std::cout << "Add " << std::endl;
			Utilities::printVectorOfFloats(gp2, 5);
		}
		start = high_resolution_clock::now();
		gp1 = gcn_layer1.backward(gp2, learningRate);
		stop = high_resolution_clock::now();
		duration = duration_cast<microseconds>(stop - start);
		if (time) {
			std::cout << "CPU : (Graph Convolution Layer 1 Backward / batch) ==> " << duration.count() << " microseconds" << std::endl;
			//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
		}

		if (debug) {
			std::cout << "GCN 1 " << std::endl;
			Utilities::printVectorOfFloats(gp1, 5);
		}
	}

	void normalize_dat(float* X, int n) {
		// Mean
		float mean[3] = { 0 };
		for (int i = 0; i < n; i++) {
			mean[0] += X[i * 3];
			mean[1] += X[i * 3 + 1];
			mean[2] += X[i * 3 + 2];
		}
		mean[0] = mean[0] / n;
		mean[1] = mean[1] / n;
		mean[2] = mean[2] / n;
		// Mean center
		for (int i = 0; i < n; i++) {
			X[i * 3] -= mean[0];
			X[i * 3 + 1] -= mean[1];
			X[i * 3 + 2] -= mean[2];
		}
		// Furthest distance
		float furthest_dist = 0;
		for (int i = 0; i < n; i++) {
			float dist = 0;
			dist = (X[i * 3] * X[i * 3]) + (X[i * 3 + 1] * X[i * 3 + 1]) + (X[i * 3 + 2] * X[i * 3 + 2]);
			dist = std::sqrt(dist);
			if (dist > furthest_dist) {
				furthest_dist = dist;
			}
		}
		// Divide by furthest distance
		for (int i = 0; i < n; i++) {
			X[i * 3] /= furthest_dist;
			X[i * 3 + 1] /= furthest_dist;
			X[i * 3 + 2] /= furthest_dist;
		}
	}

	std::vector<float*> get_laplacia(std::vector<float*> x_train) {
		std::vector<float*> laplacians;
		for (int i = 0; i < x_train.size(); i++) {
			float* current_sample = x_train[i];
			//normalize_dat(current_sample, Parameters::num_points);
			Graph::GraphCPU g(current_sample, Parameters::num_points, Parameters::input_features, Parameters::num_neighbours);
			float* L = g.get_Lnorm();
			if (debug) {
				std::cout << "Constructed graph for " << i << std::endl;
			}
			laplacians.push_back(L);
		}
		return laplacians;
	}

	void NetworkCPU::train(std::vector<float*> input, std::vector<float*> label, int n) {

		float* perEpochLoss = (float*)malloc(Parameters::num_epochs * sizeof(float));
		float epochLoss = 0;
		std::vector<float> classification;
		for (int i = 0; i < this->batchSize; i++) {
			classification.push_back(0.0f);
		}
		int num_batches = n / this->batchSize;

		// Iterate for as many epochs..
		for (int ep = 0; ep < Parameters::num_epochs; ep++) {
			std::cout << "****************************Epoch " << ep << "***************************" << std::endl;
			epochLoss = 0;

			// Loop batch by batch
			for (int b = 0; b < num_batches; b++) {
				Parameters::num_points = 1024;

				// Grab one batch's data
				std::vector<float*> batch_in = std::vector < float* >(input.begin() + b * this->batchSize, input.begin() + (b + 1) * this->batchSize);
				
				for (int bi = 0; bi < batch_in.size(); bi++) {
					normalize_dat(batch_in[bi], Parameters::num_points);
				}

				auto start = high_resolution_clock::now();
				std::vector<float*> batch_lap = get_laplacia(batch_in);
				auto stop = high_resolution_clock::now();
				auto duration = duration_cast<microseconds>(stop - start);
				if (graph_time) {
					std::cout << "************************************************************************************" << std::endl;
					std::cout << "CPU : (Graph Generation / batch) ==> " << duration.count() << " seconds" << std::endl;
					std::cout << "************************************************************************************" << std::endl;
					//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
				}

				std::vector<float*> batch;
				batch.reserve(batch_in.size() + batch_lap.size()); // preallocate memory
				batch.insert(batch.end(), batch_in.begin(), batch_in.end());
				batch.insert(batch.end(), batch_lap.begin(), batch_lap.end());
				std::vector<float*> trueLabel = std::vector < float* >(label.begin() + b * this->batchSize, label.begin() + (b + 1) * this->batchSize);
				
				
				start = high_resolution_clock::now();
				// Forward Pass
				std::vector<float*> prediction = forward(batch, false);
				stop = high_resolution_clock::now();
				duration = duration_cast<microseconds>(stop - start);
				if (time) {
					std::cout << "************************************************************************************" << std::endl;
					std::cout << "CPU : (Forward Pass / batch) ==> "<< duration.count() / 1000 << " seconds" << std::endl;
					std::cout << "************************************************************************************" << std::endl;
					//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
				}

				// Calculate Loss
				float loss = calculateLoss(prediction, trueLabel);
				epochLoss += loss;

				start = high_resolution_clock::now();
				PointCloudClassification::softmaxActivationLayerCPU softmaxLayer(numClasses, Parameters::batch_size, false);
				std::vector<float*> pprob = softmaxLayer.forward(prediction, false);
				
				// Check Prediction: Can comment this in training later on..
				getClassification(pprob, this->numClasses, classification);
				std::cout << "True Label: ";
				Utilities::printVectorOfFloats(trueLabel, Parameters::num_classes);
				// Backward Pass
				backward(pprob, trueLabel, Parameters::learning_rate);
				stop = high_resolution_clock::now();
				duration = duration_cast<microseconds>(stop - start);
				if (time) {
					std::cout << "************************************************************************************" << std::endl;
					std::cout << "CPU : (Backward Pass / batch) ==> " << duration.count() / 1000 << " seconds" << std::endl;
					std::cout << "************************************************************************************" << std::endl;
					//printElapsedTime(timer().getCpuElapsedTimeForPreviousOperation(), "CPU : (Forward Pass / batch)");
				}
			}
			epochLoss /= num_batches;
			perEpochLoss[ep] = epochLoss;
			std::cout << "Epoch: " << ep << " Loss: " << epochLoss << "\n";
		}
		std::cout << "Done with training, printing loss\n";
		Utilities::printArray(perEpochLoss, Parameters::num_epochs);

	}

	// Returns classification between [0, classes-1] for each instance
	void NetworkCPU::getClassification(const std::vector<float*> prediction, const int classes, std::vector<float> classification) {
		if (debug) {
			std::cout << "Actual Prediction: ";
			Utilities::printVectorOfFloats(prediction, Parameters::num_classes);
		}
		int n = prediction.size();
		PointCloudClassification::softmaxActivationLayerCPU softmaxLayer(numClasses, Parameters::batch_size, false);
		std::vector<float*> pprob = softmaxLayer.forward(prediction, false);
		for (int i = 0; i < n; i++) {
			float maxProb = 0;
			float clazz = 0;
			for (int j = 0; j < classes; j++) {
				if (pprob[i][j] > maxProb) {
					clazz = j;
					maxProb = pprob[i][j];
				}
			}
			classification[i] = clazz;
		}
		std::cout << "Prediction: ";
		Utilities::printVector(classification, this->batchSize);
	}
}
