#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "network.h"
#include "layers/layer.h"
#include "layers/implementations/fullyConnectedLayerGPU.cu"
#include "layers/fullyConnectedLayer.h"
#include <fstream>
#include <string>
#include ""

#ifndef imax
#define imax(a,b) (((a)>(b))?(a):(b))
#endif

#define blockSize 128

using namespace Utilities;

namespace PointCloudClassification {

	float GraphConvolutionNetworkGPU::loss(float *label, float *predicted) {

		float *devLabel;
		hipMalloc((void**)&devLabel, batchDim*layers[layers.size() - 1]->getOutputDim() * sizeof(float));
		checkCUDAError("hipMalloc");
		hipMemcpy(devLabel, label, batchDim * layers[layers.size() - 1]->getOutputDim() * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy");

		float *devPredicted;
		hipMalloc((void**)&devPredicted, batchDim*layers[layers.size() - 1]->getOutputDim() * sizeof(float));
		checkCUDAError("hipMalloc");
		hipMemcpy(devPredicted, predicted, batchDim*layers[layers.size() - 1]->getOutputDim() * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy");
		

		float *lossForEachLabel = new float[batchDim * layers[layers.size() - 1]->getOutputDim()];
		float *devLossForEachLabel;
		hipMalloc((void**)&devLossForEachLabel, batchDim * layers[layers.size() - 1]->getOutputDim() * sizeof(float));
		checkCUDAError("hipMalloc");
		

		int gridRows = ((batchDim * layers[layers.size() - 1]->getOutputDim()) + blockSize - 1) / blockSize;
		kernCrossEntropyLoss << <gridRows, blockSize >> > (batchDim * layers[layers.size() - 1]->getOutputDim(), devPredicted, devLabel, devLossForEachLabel);
		checkCUDAError("kernCrossEntropyLoss");

		hipMemcpy(lossForEachLabel, devLossForEachLabel, batchDim * layers[layers.size() - 1]->getOutputDim() * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAError("hipMemcpy");

		float loss = 0;

		for (int i = 0; i < batchDim * layers[layers.size() - 1]->getOutputDim(); i++) {
			loss += lossForEachLabel[i];
		}
		return loss / batchDim;
		
	}

	GraphConvolutionNetworkGPU::GraphConvolutionNetworkGPU(int inputDim, int numHiddenLayers, int *hiddenDim, int outputDim, int batchDim) {
		this->batchDim = batchDim;

		FullyConnectedLayer *tempLayer = new FullyConnectedLayerGPU(inputDim, hiddenDim[0], batchDim, false);
		layers.push_back(tempLayer);
		for (int i = 1; i < numHiddenLayers - 1; i++) {
			FullyConnectedLayerGPU *tempLayer = new FullyConnectedLayerGPU(hiddenDim[i - 1], hiddenDim[i], batchDim, false);
			layers.push_back(tempLayer);
		}
		tempLayer = new FullyConnectedLayerGPU(hiddenDim[numHiddenLayers - 1], outputDim, batchDim, true);
		layers.push_back(tempLayer);

	}

	void GraphConvolutionNetworkGPU::forward(float *input, float *output, bool test) {
		float *devOutput;
		hipMalloc((void**)&devOutput, batchDim * layers[0]->getInputDim() * sizeof(float));
		checkCUDAError("hipMalloc");
		hipMemcpy(devOutput, input, batchDim * layers[0]->getInputDim() * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy");
		float *hiddenOutput;
		for (int i = 0; i < layers.size(); i++) {
			hipMalloc((void**)&hiddenOutput, batchDim * layers[i]->getOutputDim() * sizeof(float));
			checkCUDAError("hipMalloc");
			layers[i]->forward(devOutput, hiddenOutput, test);
			hipFree(devOutput);
			hipMalloc((void**)&devOutput, batchDim * layers[i]->getOutputDim() * sizeof(float));
			checkCUDAError("hipMalloc");
			hipMemcpy(devOutput, hiddenOutput, batchDim * layers[i]->getOutputDim() * sizeof(float), hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy");
			hipFree(hiddenOutput);
		}
		hipMemcpy(output, devOutput, batchDim * layers[layers.size() - 1]->getOutputDim() * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAError("hipMemcpy");

		hipFree(devOutput);
	}

	void GraphConvolutionNetworkGPU::backward(float *label, float *predicted, float learningRate) {
		float *devLabel;
		hipMalloc((void**)&devLabel, batchDim*layers[layers.size() - 1]->getOutputDim() * sizeof(float));
		checkCUDAError("hipMalloc");
		hipMemcpy(devLabel, label, batchDim * layers[layers.size() - 1]->getOutputDim() * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy");

		float *devPredicted;
		hipMalloc((void**)&devPredicted, batchDim*layers[layers.size() - 1]->getOutputDim() * sizeof(float));
		checkCUDAError("hipMalloc");
		hipMemcpy(devPredicted, predicted, batchDim*layers[layers.size() - 1]->getOutputDim() * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy");

		float *incomingGradient;
		hipMalloc((void**)&incomingGradient, batchDim*layers[layers.size() - 1]->getOutputDim() * sizeof(float));
		checkCUDAError("hipMalloc");


		int gridRows = ((batchDim * layers[layers.size() - 1]->getOutputDim()) + blockSize - 1) / blockSize;
		kernSubtractMatrices << <gridRows, blockSize >> > (devPredicted, devLabel, incomingGradient, batchDim, layers[layers.size() - 1]->getOutputDim());
		checkCUDAError("kernSubtractMatrices");

		hipFree(devLabel);
		checkCUDAError("hipFree");
		hipFree(devPredicted);
		checkCUDAError("hipFree");


		checkCUDAError("hipMemcpy");
		float *outgoingGradient;
		for (int i = layers.size() - 1; i >= 0; i--) {
			hipMalloc((void**)&outgoingGradient, batchDim*layers[i]->getInputDim() * sizeof(float));
			checkCUDAError("hipMalloc");
			layers[i]->backward(incomingGradient, outgoingGradient, learningRate);
			hipFree(incomingGradient);
			checkCUDAError("hipFree");
			hipMalloc((void**)&incomingGradient, batchDim*layers[i]->getInputDim() * sizeof(float));
			checkCUDAError("hipMalloc");
			hipMemcpy(incomingGradient, outgoingGradient, batchDim*layers[i]->getInputDim() * sizeof(float), hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy");
			hipFree(outgoingGradient);
		}
		hipFree(incomingGradient);
	}


}
