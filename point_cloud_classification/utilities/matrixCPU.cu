#include "hip/hip_runtime.h"
#include "matrix.h"

/*
	A -> m x n
	B -> n x p
	output = A x B
*/
void MatrixCPU::multiply(float* A, float* B, int m, int n, int p, float* output) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < p; j++) {
			float current = 0;
			for (int k = 0; k < n; k++) {
				current += A[i * n + k] * B[k * p + j];
			}
			output[i * p + j] = current;
		}
	}
}

/*
	A -> m x n
	B -> p x n
	output = A x B.T (Multiplies A with the transpose of B)
*/
void MatrixCPU::multiplyTranspose(float* A, float* B, int m, int n, int p, float* output){
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < p; j++) {
			float current = 0;
			for (int k = 0; k < n; k++) {
				current += A[i * n + k] * B[j * n + k];
			}
			output[i * p + j] = current;
		}
	}
}

/*
	A -> m x n
	B -> p x q
	output = A (x) B (mp x nq)
*/
void MatrixCPU::kroneckerProduct(float* A, float* B, int m, int n, int p, int q, float* C) {
	/*for (int i = 0; i < (m * p); i++) {
		for (int j = 0; j < (n * q); j++) {

		}
	}*/
	std::cout << "I" << std::endl;
	for (int i = 0; i < m; i++) {
		for (int k = 0; k < p; k++) {
			for (int j = 0; j < n; j++) {
				for (int l = 0; l < q; l++) {
					
					int index = (i * n * p * q) + (k * n * q) + (j * q) + l;
					std::cout << i <<" "<< k <<" " << j <<" " << l <<" "<< std::endl;
					std::cout << A[i * n + j] << " " << B[k * q + l] <<" "<<index<< std::endl;
					C[index] = A[i * n + j] * B[k * q + l];
				}
			}
		}
	}
}

/*
	A -> m x n
	output = A.T 
*/
void MatrixCPU::transpose(float* A, int m, int n, float* output){
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			output[j * m + i] = A[i * n + j];
		}
	}
}


/*
	A -> m x n
	B -> m x n
	output = A + B
*/
void MatrixCPU::add(float* A, float* B, int m, int n, float* output) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			output[i * n + j] = A[i * n + j] + B[i * n + j];
		}
	}
}

/*
	A -> m x n
	B -> m x n
	output = A - B
*/
void MatrixCPU::subtract(float* A, float* B, int m, int n, float* output) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			output[i * n + j] = A[i * n + j] - B[i * n + j];
		}
	}
}

/*
	A -> m x m
	output = A - I
*/
void MatrixCPU::subtractIdentity(float* A, int m) {
	for (int i = 0; i < m; i++) {
		A[i * m + i] = A[i * m + i] - 1;
	}
}

/*
	A -> m x n
	B -> m x n
	output = A - alpha * B
*/
void MatrixCPU::subtractWithFactor(float* A, float* B, float alpha, int m, int n, float* output) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			output[i * n + j] = A[i * n + j] - (alpha * B[i * n + j]);
		}
	}
}

/*
	A -> m x n
	B -> m x n
	output = alpha * A + beta * B
*/
void MatrixCPU::linearCombination(float* A, float* B, float alpha, float beta, int m, int n, float* output) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			output[i * n + j] = (alpha * A[i * n + j]) + (beta * B[i * n + j]);
		}
	}
}

/*
	A -> m x n
	output = n
*/
void MatrixCPU::maxAcrossDim1(float* A, int m, int n, int* argmaxOutput, float* output) {
	for (int i = 0; i < n; i++) {
		output[i] = FLT_MIN;
		for (int j = 0; j < m; j++) {
			if (A[n*j + i] > output[i]) {
				output[i] = A[n*j + i];
				argmaxOutput[i] = j;
			}
		}
	}
}

/*
	A -> m x n
	output = n
*/
void MatrixCPU::meanAcrossDim1(float* A, int m, int n, float* output) {
	for (int i = 0; i < n; i++) {
		output[i] = 0;
		for (int j = 0; j < m; j++) {
			output[i] += A[n*j + i];
		}
		output[i] /= m;
	}
}

/*
	A -> m x n
	output = n
*/
void MatrixCPU::varianceAcrossDim1(float* A, int m, int n, float* output, float* mean) {

	for (int i = 0; i < n; i++) {
		output[i] = 0;
		for (int j = 0; j < m; j++) {
			output[i] += std::pow((A[n*j + i] - mean[i]), 2);
		}
		output[i] /= m;
	}
}

/*
	Prints matrix A
*/
void MatrixCPU::printMatrix(float* A, int m, int n) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			printf("%f ", A[i * n + j]);
		}
		printf("\n");
	}
}

/*
	returns Identity matrix of given dimension
*/
void MatrixCPU::getIdentityMatrix(int m, float* A) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < m; j++) {
			if (i == j){
				A[i * m + j] = 1;
			}else{
				A[i * m + j] = 0;
			}
		}
	}
}


